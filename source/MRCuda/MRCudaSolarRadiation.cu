#include "hip/hip_runtime.h"
#include "MRCudaSolarRadiation.cuh"
#include "MRCudaBasic.h"
#include ""
#include "MRCudaFloat.cuh"
#include <float.h>

namespace MR
{
namespace Cuda
{

__global__ void rayKernel( const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces, const float3* samples, const uint64_t* validSamples, const IntersectionPrecomputes* precs, uint64_t* res, const size_t resBlockCount, const size_t sampleCount, const size_t precCount, MeshIntersectionResult* outIntersections )
{
    if ( resBlockCount == 0 )
    {
        assert( false );
        return;
    }

    const size_t blockIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if ( blockIndex >= resBlockCount )
        return;

    const size_t blockStart = ( blockIndex << 6 ); // use bit shift instead of multiplying on 64
    const size_t blockEnd = ( blockIndex == resBlockCount - 1) ? ( sampleCount * precCount ) : ( ( blockIndex + 1 ) << 6 );

    uint64_t currentBit = 1;
    uint64_t block = 0;
    for ( size_t index = blockStart; index < blockEnd; ++index )
    {
        const size_t sample = index / precCount;
        if ( testBit( validSamples, sample ) )
        {
            const size_t patch = index % precCount;
            const auto intersectRes = rayMeshIntersect( nodes, meshPoints, faces, samples[sample], 0, FLT_MAX, precs[patch], bool( outIntersections ) );
            if ( intersectRes.distanceAlongLine > 0 )
            {
                block |= currentBit;         
            }
            else if ( outIntersections )
            {
                outIntersections[index] = intersectRes;
            }
        }
        currentBit <<= 1;
    }

    res[blockIndex] = block;
}

__global__ void radiationKernel( const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces, const float3* samples, const uint64_t* validSamples, const SkyPatch* skyPatches, const IntersectionPrecomputes* precs, const float rMaxRadiation, float* res, const size_t sampleCount, const size_t precCount, MeshIntersectionResult* outIntersections )
{
    if ( sampleCount == 0 )
    {
        assert( false );
        return;
    }

    const size_t sampleVertId = blockIdx.x * blockDim.x + threadIdx.x;
    if ( sampleVertId >= sampleCount || !testBit( validSamples, sampleVertId ) )
        return;

    const auto samplePt = samples[sampleVertId];

    float totalRadiation = 0;
    for ( int i = 0; i < precCount; ++i )
    {
        const auto intersectRes = rayMeshIntersect( nodes, meshPoints, faces, samplePt, 0, FLT_MAX, precs[i], bool( outIntersections ) );
        if ( intersectRes.distanceAlongLine < 0 )
            totalRadiation += skyPatches[i].radiation;
        else if ( outIntersections )
            outIntersections[sampleVertId * precCount + i] = intersectRes;

    }

    res[sampleVertId] = rMaxRadiation * totalRadiation;
}

__global__ void radiationKernel( const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces, const float3* samples, const uint64_t* validSamples, const SkyPatch* skyPatches, const IntersectionPrecomputes* precs, const float rMaxRadiation, float* res, const size_t sampleCount, const size_t precCount, uint64_t* outSkyRays )
{
    if ( sampleCount == 0 )
    {
        assert( false );
        return;
    }

    const size_t sampleVertId = blockIdx.x * blockDim.x + threadIdx.x;
    if ( sampleVertId >= sampleCount || !testBit( validSamples, sampleVertId ) )
        return;
    
    float totalRadiation = 0;
    auto ray = size_t( sampleVertId ) * precCount;
    for ( int i = 0; i < precCount; ++i, ++ray )
    {
        if ( testBit( outSkyRays, ray ) )
            totalRadiation += skyPatches[i].radiation;
    }

    res[sampleVertId] = rMaxRadiation * totalRadiation;
}

hipError_t findSkyRaysKernel( const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces, const float3* samples, const uint64_t* validSamples, const IntersectionPrecomputes* precs, uint64_t* res, const size_t resBlockCount, const size_t sampleCount, const size_t precCount, MeshIntersectionResult* outIntersections )
{
    constexpr int maxThreadsPerBlock = 640;
    int numBlocks = ( int( resBlockCount ) + maxThreadsPerBlock - 1 ) / maxThreadsPerBlock;

    rayKernel << < numBlocks, maxThreadsPerBlock >> > ( nodes, meshPoints, faces, samples, validSamples, precs, res, resBlockCount, sampleCount, precCount, outIntersections );
    CUDA_EXEC_RETURN( hipGetLastError() );

    return hipSuccess;
}

hipError_t computeSkyViewFactorKernel( const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces, const float3* samples, const uint64_t* validSamples, const SkyPatch* skyPatches, const IntersectionPrecomputes* precs, const float rMaxRadiation, float* res, const size_t sampleCount, const size_t precCount, MeshIntersectionResult* outIntersections )
{
    constexpr int maxThreadsPerBlock = 640;
    int numBlocks = (int( sampleCount ) + maxThreadsPerBlock - 1) / maxThreadsPerBlock;
   
    radiationKernel << < numBlocks, maxThreadsPerBlock >> > (nodes, meshPoints, faces, samples, validSamples, skyPatches, precs, rMaxRadiation, res, sampleCount, precCount, outIntersections );

    CUDA_EXEC_RETURN( hipGetLastError() );

    return hipSuccess;
}

hipError_t computeSkyViewFactorKernel( const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces, const float3* samples, const uint64_t* validSamples, const SkyPatch* skyPatches, const IntersectionPrecomputes* precs, const float rMaxRadiation, float* res, const size_t sampleCount, const size_t precCount, uint64_t* outSkyRays )
{
    constexpr int maxThreadsPerBlock = 640;
    int numBlocks = ( int( sampleCount ) + maxThreadsPerBlock - 1 ) / maxThreadsPerBlock;
    radiationKernel << < numBlocks, maxThreadsPerBlock >> > ( nodes, meshPoints, faces, samples, validSamples, skyPatches, precs, rMaxRadiation, res, sampleCount, precCount, outSkyRays );
    CUDA_EXEC_RETURN( hipGetLastError() );

    return hipSuccess;
}

}
}