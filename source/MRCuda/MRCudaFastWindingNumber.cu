#include "hip/hip_runtime.h"
#include "MRCudaFastWindingNumber.cuh"
#include "MRMesh/MRAABBTree.h"
#include "MRMesh/MRConstants.h"
#include ""

namespace MR
{

namespace Cuda
{

constexpr int maxThreadsPerBlock = 32;

__device__ float triangleSolidAngle( const float3& p, const float3& tri0, const float3& tri1, const float3& tri2 )
{
    const auto mx = tri0 - p;
    const auto my = tri1 - p;
    const auto mz = tri2 - p;

    const auto x = length( mx );
    const auto y = length( my );
    const auto z = length( mz );

    auto den = x * y * z + dot( mx, my ) * z + dot( my, mz ) * x + dot( mz, mx ) * y;
    return 2 * std::atan2( mx.x * ( my.y * mz.z - my.z * mz.y ) - mx.y * ( my.x * mz.z - my.z * mz.x ) + mx.z * ( my.x * mz.y - my.y * mz.x ), den );
}

__device__ void processPoint( const float3& q, float& res, const Dipole* dipoles,
    const Node3* __restrict__ nodes, const float3* __restrict__ meshPoints, const FaceToThreeVerts* __restrict__ faces,
    float beta, int skipFace = -1 )
{
    const float betaSq = beta * beta;
    constexpr int MaxStackSize = 32; // to avoid allocations
    int subtasks[MaxStackSize];
    int stackSize = 0;
    subtasks[stackSize++] = 0;

    while ( stackSize > 0 )
    {
        const auto i = subtasks[--stackSize];
        const auto& node = nodes[i];
        const auto& d = dipoles[i];
        if ( d.addIfGoodApprox( q, betaSq, res ) )
            continue;
        if ( !node.leaf() )
        {
            // recurse deeper
            subtasks[stackSize++] = node.r; // to look later
            subtasks[stackSize++] = node.l; // to look first
            continue;
        }
        if ( node.leafId() != skipFace )
        {
            const auto faceVerts = faces[node.leafId()];
            res += triangleSolidAngle( q, meshPoints[faceVerts.verts[0]], meshPoints[faceVerts.verts[1]], meshPoints[faceVerts.verts[2]] );
        }
    }
    constexpr float INV_4PI = 1.0f / ( 4 * PI_F );
    res *= INV_4PI;
}

__device__ float calcDistance( const float3& pt,
    const Node3* __restrict__ nodes, const float3* __restrict__ meshPoints, const FaceToThreeVerts* __restrict__ faces,
    float maxDistSq, float minDistSq )
{
    float resSq = maxDistSq;
    struct SubTask
    {
        int n;
        float distSq;
    };

    constexpr int MaxStackSize = 32; // to avoid allocations
    SubTask subtasks[MaxStackSize];
    int stackSize = 0;

    auto addSubTask = [&] ( const SubTask& s )
    {
        if ( s.distSq < resSq )
        {
            assert( stackSize < MaxStackSize );
            subtasks[stackSize++] = s;
        }
    };

    auto getSubTask = [&] ( int n )
    {
        const auto box = nodes[n].box;
        float distSq = lengthSq( box.getBoxClosestPointTo( pt ) - pt );
        return SubTask{ n, distSq };
    };

    addSubTask( getSubTask( 0 ) );

    while ( stackSize > 0 )
    {
        const auto s = subtasks[--stackSize];
        const auto& node = nodes[s.n];
        if ( s.distSq >= resSq )
            continue;

        if ( node.leaf() )
        {
            const auto face = node.leafId();
            const auto& vs = faces[face].verts;
            float3 a = meshPoints[vs[0]];
            float3 b = meshPoints[vs[1]];
            float3 c = meshPoints[vs[2]];

            // compute the closest point in double-precision, because float might be not enough
            const auto closestPointRes = closestPointInTriangle( pt, a, b, c );

            float distSq = lengthSq( closestPointRes.proj - pt );
            if ( distSq < resSq )
                resSq = distSq;
            if ( distSq <= minDistSq )
                break;
            continue;
        }

        auto s1 = getSubTask( node.l );
        auto s2 = getSubTask( node.r );
        if ( s1.distSq < s2.distSq )
        {
            const auto temp = s1;
            s1 = s2;
            s2 = temp;
        }
        assert( s1.distSq >= s2.distSq );
        addSubTask( s1 ); // larger distance to look later
        addSubTask( s2 ); // smaller distance to look first
    }
    return sqrt( resSq );
}

__global__ void fastWindingNumberFromVectorKernel( const float3* points,
    const Dipole* __restrict__ dipoles, const Node3* __restrict__ nodes, const float3* __restrict__ meshPoints, const FaceToThreeVerts* __restrict__ faces,
    float* resVec, float beta, int skipFace, size_t size )
{
    if ( size == 0 )
    {
        assert( false );
        return;
    }

    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if ( index >= size )
        return;

    processPoint( points[index], resVec[index], dipoles, nodes, meshPoints, faces, beta, skipFace );
}

__global__ void fastWindingNumberFromMeshKernel( const Dipole* __restrict__ dipoles,
    const Node3* __restrict__ nodes, const float3* __restrict__ meshPoints, const FaceToThreeVerts* __restrict__ faces,
    float* resVec, float beta, size_t size )
{
    if ( size == 0 )
    {
        assert( false );
        return;
    }

    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if ( index >= size )
        return;        

    const auto& face = faces[index];
    if ( face.verts[0] < 0 || face.verts[1] < 0 || face.verts[2] < 0 )
        return;

    const auto q = ( meshPoints[face.verts[0]] + meshPoints[face.verts[1]] + meshPoints[face.verts[2]] ) / 3.0f;
    processPoint( q, resVec[index], dipoles, nodes, meshPoints, faces, beta, index );
}

__global__ void fastWindingNumberFromGridKernel( int3 dims, Matrix4 gridToMeshXf,
    const Dipole* __restrict__ dipoles, const Node3* __restrict__ nodes, const float3* __restrict__ meshPoints, const FaceToThreeVerts* __restrict__ faces,
    float* resVec, float beta, size_t size )
{
    if ( size == 0 )
    {
        assert( false );
        return;
    }

    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if ( index >= size )
        return;

    const int sizeXY = dims.x * dims.y;
    const int sumZ = int( index % sizeXY );
    const int3 voxel{ sumZ % dims.x, sumZ / dims.x, int( index / sizeXY ) };
    const float3 point{ float( voxel.x ), float( voxel.y ), float( voxel.z ) };
    const float3 transformedPoint = gridToMeshXf.isIdentity ? point : gridToMeshXf.transform( point );

    processPoint( transformedPoint, resVec[index], dipoles, nodes, meshPoints, faces, beta, index );
}

__global__ void signedDistanceKernel( int3 dims, Matrix4 gridToMeshXf,
    const Dipole* __restrict__ dipoles, const Node3* __restrict__ nodes, const float3* __restrict__ meshPoints, const FaceToThreeVerts* __restrict__ faces,
    float* resVec, float windingNumberThreshold, float beta, float maxDistSq, float minDistSq, size_t size )
{
    if ( size == 0 )
    {
        assert( false );
        return;
    }

    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if ( index >= size )
        return;

    const int sizeXY = dims.x * dims.y;
    const int sumZ = int( index % sizeXY );
    const int3 voxel{ sumZ % dims.x, sumZ / dims.x, int( index / sizeXY ) };
    const float3 point{ float( voxel.x ), float( voxel.y ), float( voxel.z ) };
    const float3 transformedPoint = gridToMeshXf.isIdentity ? point : gridToMeshXf.transform( point );

    float& res = resVec[index];
    res = calcDistance( transformedPoint, nodes, meshPoints, faces, maxDistSq, minDistSq );

    float fwn{ 0 };
    processPoint( transformedPoint, fwn, dipoles, nodes, meshPoints, faces, beta, index );
    if ( fwn > windingNumberThreshold )
        res = -res;
}

void fastWindingNumberFromVector( const float3* points, const Dipole* dipoles,
                                const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces,
                                float* resVec, float beta, int skipFace, size_t size )
{
    int numBlocks = ( int( size ) + maxThreadsPerBlock - 1 ) / maxThreadsPerBlock;
    fastWindingNumberFromVectorKernel<<< numBlocks, maxThreadsPerBlock >>>( points, dipoles, nodes, meshPoints, faces, resVec, beta, skipFace, size );
}

void fastWindingNumberFromMesh( const Dipole* dipoles,
                                        const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces,
                                        float* resVec, float beta, size_t size )
{
    int numBlocks = ( int( size ) + maxThreadsPerBlock - 1 ) / maxThreadsPerBlock;
    fastWindingNumberFromMeshKernel<<< numBlocks, maxThreadsPerBlock >>>( dipoles, nodes, meshPoints, faces, resVec, beta, size );
}

void fastWindingNumberFromGrid( int3 dims, Matrix4 gridToMeshXf,
                                        const Dipole* dipoles, const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces,
                                        float* resVec, float beta )
{
    const size_t size = size_t( dims.x ) * dims.y * dims.z;
    int numBlocks = ( int( size ) + maxThreadsPerBlock - 1 ) / maxThreadsPerBlock;
    fastWindingNumberFromGridKernel<<< numBlocks, maxThreadsPerBlock >>>( dims, gridToMeshXf, dipoles, nodes, meshPoints, faces, resVec, beta, size );       
}

void signedDistance( int3 dims, Matrix4 gridToMeshXf,
                                        const Dipole* dipoles, const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces,
                                        float* resVec, float windingNumberThreshold, float beta, float maxDistSq, float minDistSq )
{
    const size_t size = size_t( dims.x ) * dims.y * dims.z;
    int numBlocks = ( int( size ) + maxThreadsPerBlock - 1 ) / maxThreadsPerBlock;
    signedDistanceKernel<<< numBlocks, maxThreadsPerBlock >>>( dims, gridToMeshXf, dipoles, nodes, meshPoints, faces, resVec, windingNumberThreshold, beta, maxDistSq, minDistSq, size );
}

} //namespece Cuda

} //namespace MR
