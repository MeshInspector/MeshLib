#include "hip/hip_runtime.h"
#include "MRCudaPointsProject.cuh"

#include ""

namespace MR::Cuda
{

__global__ void kernel( PointsProjectionResult* __restrict__ res, PointCloudData pc, const float3* __restrict__ points,
    Matrix4 pointsXf, Matrix4 refXf, float upDistLimitSq, float loDistLimitSq, size_t chunkSize, size_t chunkOffset )
{
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    if ( index >= chunkSize )
        return;

    const auto pt = pointsXf.isIdentity ? points[index] : pointsXf.transform( points[index] );

    PointsProjectionResult result {
        .distSq = upDistLimitSq,
        .vertId = -1,
    };

    struct SubTask
    {
        int n;
        float distSq;
    };

    constexpr int MaxStackSize = 32; // to avoid allocations
    SubTask subtasks[MaxStackSize];
    int stackSize = 0;

    auto addSubTask = [&] ( const SubTask& s )
    {
        if ( s.distSq < result.distSq )
        {
            assert( stackSize < MaxStackSize );
            subtasks[stackSize++] = s;
        }
    };

    auto getSubTask = [&] ( int n )
    {
        const auto box = pc.nodes[n].box;
        const auto distSq = lengthSq( box.getBoxClosestPointTo( pt ) - pt );
        return SubTask{ n, distSq };
    };

    addSubTask( getSubTask( 0 ) );

    while ( stackSize > 0 )
    {
        const auto s = subtasks[--stackSize];
        const auto& node = pc.nodes[s.n];
        if ( s.distSq >= result.distSq )
            continue;

        if ( node.leaf() )
        {
            auto [begin, end] = node.getLeafPointRange();
            for ( int i = begin; i < end; ++i )
            {
                const auto proj = refXf.isIdentity ? pc.points[i].coord : refXf.transform( pc.points[i].coord );
                const auto distSq = lengthSq( proj - pt );
                if ( distSq < result.distSq )
                {
                    result = { distSq, pc.points[i].id };
                    if ( distSq <= loDistLimitSq )
                        goto exit;
                }
            }
        }

        auto s1 = getSubTask( node.l );
        auto s2 = getSubTask( node.r );
        if ( s1.distSq < s2.distSq )
        {
            const auto temp = s1;
            s1 = s2;
            s2 = temp;
        }
        assert( s1.distSq >= s2.distSq );
        addSubTask( s1 ); // larger distance to look later
        addSubTask( s2 ); // smaller distance to look first
    }

exit:
    res[index] = result;
}

void findProjectionOnPointsKernel( PointsProjectionResult* res, PointCloudData pc, const float3* points,
    Matrix4 pointsXf, Matrix4 refXf, float upDistLimitSq, float loDistLimitSq, size_t chunkSize, size_t chunkOffset )
{
    constexpr int maxThreadsPerBlock = 640;
    const auto numBlocks = (int)( ( chunkSize + maxThreadsPerBlock - 1 ) / maxThreadsPerBlock );
    kernel <<< numBlocks, maxThreadsPerBlock >>> ( res, pc, points, pointsXf, refXf, upDistLimitSq, loDistLimitSq, chunkSize, chunkOffset );
}

} // namespace MR::Cuda
