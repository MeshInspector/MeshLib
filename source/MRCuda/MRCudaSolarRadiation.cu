#include "hip/hip_runtime.h"
#include "MRCudaSolarRadiation.cuh"
#include "MRCudaBasic.h"
#include ""
#include "MRCudaFloat.cuh"
#include <float.h>

namespace MR
{
namespace Cuda
{



__global__ void rayKernel( const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces, const float3* samples, const uint64_t* validSamples, const IntersectionPrecomputes* precs, uint64_t* res, const size_t sampleCount, const size_t precCount, int& trueBitCount )
{
    const size_t size = sampleCount * precCount;
    if ( size == 0 )
    {
        assert( false );
        return;
    }

    const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if ( index >= 50000 )
        return;

    const size_t sample = index / precCount;
    if ( !testBit( validSamples, sample ) )
        return;

    const size_t patch = index % precCount;

    if ( rayMeshIntersect( nodes, meshPoints, faces, samples[sample], 0, FLT_MAX, precs[patch] ) < 0 )
    {
        setBit( res, index );
        //++trueBitCount;
    }
}

__global__ void radiationKernel( const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces, const float3* samples, const uint64_t* validSamples, const SkyPatch* skyPatches, const IntersectionPrecomputes* precs, const float rMaxRadiation, float* res, const size_t sampleCount, const size_t precCount, uint64_t* outSkyRays )
{
    if ( sampleCount == 0 )
    {
        assert( false );
        return;
    }

    const size_t sampleVertId = blockIdx.x * blockDim.x + threadIdx.x;
    if ( sampleVertId >= sampleCount )
        return;

    if ( outSkyRays )
    {
        float totalRadiation = 0;
        auto ray = size_t( sampleVertId ) * precCount;
        for ( int i = 0; i < precCount; ++i, ++ray )
        {
            if ( testBit( outSkyRays, ray ) )
                totalRadiation += skyPatches[i].radiation;
        }

        res[sampleVertId] = rMaxRadiation * totalRadiation;
        return;
    }

    const auto samplePt = samples[sampleVertId];

    float totalRadiation = 0;
    for ( int i = 0; i < precCount; ++i )
    {
        if ( rayMeshIntersect( nodes, meshPoints, faces, samplePt, 0, FLT_MAX, precs[i] ) < 0 )
            totalRadiation += skyPatches[i].radiation;
    }

    res[sampleVertId] = rMaxRadiation * totalRadiation;
}

void findSkyRaysKernel( const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces, const float3* samples, const uint64_t* validSamples, const IntersectionPrecomputes* precs, uint64_t* res, const size_t sampleCount, const size_t precCount, int& trueBitCount )
{
    const size_t resSize = sampleCount * precCount;
    constexpr int maxThreadsPerBlock = 640;
    int numBlocks = ( int( resSize ) + maxThreadsPerBlock - 1 ) / maxThreadsPerBlock;

    rayKernel << < numBlocks, maxThreadsPerBlock >> > ( nodes, meshPoints, faces, samples, validSamples, precs, res, sampleCount, precCount, trueBitCount );
}

void computeSkyViewFactorKernel( const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces, const float3* samples, const uint64_t* validSamples, const SkyPatch* skyPatches, const IntersectionPrecomputes* precs, const float rMaxRadiation, float* res, const size_t sampleCount, const size_t precCount, uint64_t* outSkyRays )
{
    constexpr int maxThreadsPerBlock = 640;
    int numBlocks = ( int( sampleCount ) + maxThreadsPerBlock - 1 ) / maxThreadsPerBlock;
    radiationKernel << < numBlocks, maxThreadsPerBlock >> > ( nodes, meshPoints, faces, samples, validSamples, skyPatches, precs, rMaxRadiation, res, sampleCount, precCount, outSkyRays );
}

}
}