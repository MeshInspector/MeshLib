#include "hip/hip_runtime.h"
﻿#include "MRCudaTest.h"
#include "MRMesh/MRImage.h"
#include "hip/hip_runtime.h"
#include ""


namespace MR
{

namespace Cuda
{

__global__ void negateKernel( uint8_t* imagePtr, const int size )
{
    int pixelShift = blockIdx.x * blockDim.x + threadIdx.x;
    if ( pixelShift >= size )
        return;

    int ind = 4 * pixelShift;

    imagePtr[ind] = 255 - imagePtr[ind];
    imagePtr[ind + 1] = 255 - imagePtr[ind + 1];
    imagePtr[ind + 2] = 255 - imagePtr[ind + 2];
}

void negatePicture( Image& image )
{
    hipSetDevice( 0 );
    uint8_t* cudaPointer{ nullptr };
    auto size = image.resolution.x * image.resolution.y;
    hipMalloc( ( void** )&cudaPointer, size * sizeof( uint8_t ) * 4 );
    hipMemcpy( cudaPointer, image.pixels.data(), size * sizeof( uint8_t ) * 4, hipMemcpyHostToDevice );
    int maxThreadsPerBlock = 0;
    hipDeviceGetAttribute( &maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0 );
    int numBlocks = ( size + maxThreadsPerBlock - 1 ) / maxThreadsPerBlock;
    // kernel
    negateKernel << <numBlocks, maxThreadsPerBlock >> > ( cudaPointer, size );

    hipMemcpy( image.pixels.data(), cudaPointer, size * sizeof( uint8_t ) * 4, hipMemcpyDeviceToHost );
    hipFree( cudaPointer );
}
}
}