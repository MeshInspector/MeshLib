#include "hip/hip_runtime.h"
#include "MRCudaPointsProject.cuh"

#include ""

namespace MR::Cuda
{

__global__ void kernel( PointsProjectionResult* __restrict__ res, PointCloudData pc, const float3* __restrict__ points,
    const uint64_t* __restrict__ validPoints, Matrix4 xf, float upDistLimitSq, float loDistLimitSq, bool skipSameIndex,
    size_t chunkSize, size_t chunkOffset )
{
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    if ( index >= chunkSize )
        return;

    const auto globalIndex = index + chunkOffset;
    if ( validPoints && !testBit( validPoints, globalIndex ) )
        return;

    const auto pt = xf.isIdentity ? points[index] : xf.transform( points[index] );

    PointsProjectionResult result;
    result.distSq = upDistLimitSq;
    result.vertId = -1;

    struct SubTask
    {
        int n;
        float distSq;
    };

    constexpr int MaxStackSize = 32; // to avoid allocations
    SubTask subtasks[MaxStackSize];
    int stackSize = 0;

    auto addSubTask = [&] ( const SubTask& s )
    {
        if ( s.distSq < result.distSq )
        {
            assert( stackSize < MaxStackSize );
            subtasks[stackSize++] = s;
        }
    };

    auto getSubTask = [&] ( int n )
    {
        const auto box = pc.nodes[n].box;
        const auto distSq = lengthSq( box.getBoxClosestPointTo( pt ) - pt );
        return SubTask{ n, distSq };
    };

    addSubTask( getSubTask( 0 ) );

    while ( stackSize > 0 )
    {
        const auto s = subtasks[--stackSize];
        const auto& node = pc.nodes[s.n];
        if ( s.distSq >= result.distSq )
            continue;

        if ( node.leaf() )
        {
            auto [begin, end] = node.getLeafPointRange();
            for ( int i = begin; i < end; ++i )
            {
                if ( skipSameIndex && i == globalIndex )
                    continue;

                const auto proj = pc.points[i].coord;
                const auto distSq = lengthSq( proj - pt );
                if ( distSq < result.distSq )
                {
                    result = { distSq, pc.points[i].id };
                    if ( distSq <= loDistLimitSq )
                        goto exit;
                }
            }
            continue;
        }

        auto s1 = getSubTask( node.l );
        auto s2 = getSubTask( node.r );
        if ( s1.distSq < s2.distSq )
        {
            const auto temp = s1;
            s1 = s2;
            s2 = temp;
        }
        assert( s1.distSq >= s2.distSq );
        addSubTask( s1 ); // larger distance to look later
        addSubTask( s2 ); // smaller distance to look first
    }

exit:
    res[index] = result;
}

void findProjectionOnPointsKernel( PointsProjectionResult* res, PointCloudData pc, const float3* points,
    const uint64_t* validPoints, Matrix4 xf, float upDistLimitSq, float loDistLimitSq, bool skipSameIndex,
    size_t chunkSize, size_t chunkOffset )
{
    constexpr int maxThreadsPerBlock = 640;
    const auto numBlocks = (int)( ( chunkSize + maxThreadsPerBlock - 1 ) / maxThreadsPerBlock );
    kernel <<< numBlocks, maxThreadsPerBlock >>> ( res, pc, points, validPoints, xf, upDistLimitSq, loDistLimitSq, skipSameIndex, chunkSize, chunkOffset );
}

} // namespace MR::Cuda
