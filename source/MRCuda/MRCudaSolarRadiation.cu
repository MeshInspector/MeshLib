#include "hip/hip_runtime.h"
#include "MRCudaSolarRadiation.cuh"
#include "MRCudaBasic.h"
#include ""
#include "MRCudaFloat.cuh"
#include <float.h>

namespace MR
{
namespace Cuda
{

__device__ bool testBit( const uint64_t* bitSet, size_t bitNumber )
{
    return bool( ( bitSet[bitNumber / 64] >> ( bitNumber % 64 ) ) & 1 );
}

__device__ bool setBit( uint64_t* bitSet, size_t bitNumber )
{
    bitSet[bitNumber / 64] |= ( 1 << ( bitNumber % 64 ) );
}

__global__ void kernel( const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces, const float3* samples, const uint64_t* validSamples, const IntersectionPrecomputes* precs, uint64_t* res, const size_t nodeCount, const size_t sampleCount, const size_t precCount )
{
    const size_t size = sampleCount * precCount;
    if ( size == 0 )
    {
        assert( false );
        return;
    }

    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if ( index >= size )
        return;

    const size_t sample = index / precCount;
    if ( !testBit( validSamples, sample ) )
        return;

    const size_t patch = index % precCount;

    if ( rayMeshIntersect( nodes, meshPoints, faces, samples[sample], 0, FLT_MAX, precs[patch] ) > 0 )
        setBit( res, index );
}

__global__ void radiationKernel( const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces, const float3* samples, const uint64_t* validSamples, const SkyPatch* skyPatches, const IntersectionPrecomputes* precs, const float rMaxRadiation, float* res, const size_t nodeCount, const size_t sampleCount, const size_t precCount, uint64_t* outSkyRays )
{
    if ( sampleCount == 0 )
    {
        assert( false );
        return;
    }

    size_t sampleVertId = blockIdx.x * blockDim.x + threadIdx.x;
    if ( sampleVertId >= sampleCount )
        return;

    if ( outSkyRays )
    {
        float totalRadiation = 0;
        auto ray = size_t( sampleVertId ) * precCount;
        for ( int i = 0; i < precCount; ++i, ++ray )
        {
            if ( testBit( outSkyRays, ray ) )
                totalRadiation += skyPatches[i].radiation;
        }

        res[sampleVertId] = rMaxRadiation * totalRadiation;
        return;
    }

    const auto samplePt = samples[sampleVertId];

    float totalRadiation = 0;
    for ( int i = 0; i < precCount; ++i )
    {
        if ( !rayMeshIntersect( nodes, meshPoints, faces, samplePt, 0, FLT_MAX, precs[i] ) )
            totalRadiation += skyPatches[i].radiation;
    }

    res[sampleVertId] = rMaxRadiation * totalRadiation;
}

void findSkyRaysKernel( const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces, const float3* samples, const uint64_t* validSamples, const IntersectionPrecomputes* precs, uint64_t* res, const size_t nodeCount, const size_t sampleCount, const size_t precCount )
{
    const size_t resSize = sampleCount * precCount;
    constexpr int maxThreadsPerBlock = 640;
    int numBlocks = ( int( resSize ) + maxThreadsPerBlock - 1 ) / maxThreadsPerBlock;

    kernel << < numBlocks, maxThreadsPerBlock >> > ( nodes, meshPoints, faces, samples, validSamples, precs, res, nodeCount, sampleCount, precCount );
}

void computeSkyViewFactorKernel( const Node3* nodes, const float3* meshPoints, const FaceToThreeVerts* faces, const float3* samples, const uint64_t* validSamples, const SkyPatch* skyPatches, const IntersectionPrecomputes* precs, const float rMaxRadiation, float* res, const size_t nodeCount, const size_t sampleCount, const size_t precCount, uint64_t* outSkyRays )
{
    constexpr int maxThreadsPerBlock = 640;

    if ( outSkyRays )
        findSkyRaysKernel( nodes, meshPoints, faces, samples, validSamples, precs, outSkyRays, nodeCount, sampleCount, precCount );

    int numBlocks = ( int( sampleCount ) + maxThreadsPerBlock - 1 ) / maxThreadsPerBlock;
    radiationKernel << < numBlocks, maxThreadsPerBlock >> > ( nodes, meshPoints, faces, samples, validSamples, skyPatches, precs, rMaxRadiation, res, nodeCount, sampleCount, precCount, outSkyRays );
}

}
}